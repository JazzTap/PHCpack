#ifndef CUDA_SET_CU_
#define CUDA_SET_CU_

#include "cuda_set.h"

void cuda_set()
{
   hipSetDevice(0);
   if(hipSuccess
      != hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte))
   {
      std::cout << "Error setting CUDA device!\n" << std::endl;
   }
/*        report only the error when setting CUDA device
   else
   {
      std::cout << "Successfully set CUDA device!\n" << std::endl;
   }
*/
   hipDeviceSetCacheConfig(hipFuncCachePreferShared);
}

#endif /* CUDA_SET_CU_ */
